#include "hip/hip_runtime.h"
#include "stdio.h"
#include "common/book.h"

// global will let the compiler know that this should run on device
// instead of host
__global__ void add(int a, int b, int *c) {
  *c = a + b;
}

int main() {
  int c;
  int *dev_c;
  int a = 2, b = 7;
  HANDLE_ERROR(hipMalloc((void**) &dev_c, sizeof(int))); // allocated on device
  add<<<1, 1>>>(a, b, dev_c);
  // copy result from device to host
  HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
  printf("%d + %d = %d\n", a, b, c);
  hipFree(dev_c);
  return 0;
}
