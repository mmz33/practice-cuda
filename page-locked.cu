#include <stdio.h>
#include "../common/book.h"

/*
 * Tested on GPU GTX 980
 * Interesting results below:
  
  During copy from host to device
  Time using pageable memory: 1307.5 ms
  MB/s: 3059.3

  During copy from device to host
  Time using pageable memory: 1317.7 ms
  MB/s: 3035.6

  During copy from host to device
  Time using page-locked memory: 706.6 ms
  MB/s: 5661.0

  During copy from device to host
  Time using page-locked memory: 630.1 ms
  MB/s: 6348.0
*/


#define SIZE (10 * 1024 * 1024) // 10 MB

float cuda_host_alloc_test(int size, bool pageable, bool host_to_device) {
  hipEvent_t start, stop;
  int *a, *dev_a;
  float elapsed_time;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));

  if (pageable) {
    a = (int*)malloc(size * sizeof(int));
  } else {
    HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(int), hipHostMallocDefault));
  }

  HANDLE_NULL(a);
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size * sizeof(int)));

  HANDLE_ERROR(hipEventRecord(start, 0));
  // 100 copies
  for (int i = 0; i < 100; ++i) {
    if (host_to_device) {
      HANDLE_ERROR(hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice));
    } else {
      HANDLE_ERROR(hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost));
    }
  }
  HANDLE_ERROR(hipEventRecord(stop, 0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, stop));
  
  if (pageable)
    free(a);
  else
    HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  return elapsed_time;
}

void print_time(float MB, float elapsed_time, bool pageable, bool host_to_device) {
  const char* direction;
  if (host_to_device)
    direction = "host to device";
  else
    direction = "device to host";

  const char* mem_type;
  if (pageable)
    mem_type = "pageable";
  else
    mem_type = "page-locked";
  
  printf("During copy from %s\n", direction);
  printf("Time using %s memory: %3.1f ms\n", mem_type, elapsed_time);
  printf("MB/s: %3.1f\n\n", MB/(elapsed_time/1000));
}
  
int main(void) {
  float elapsed_time;
  float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;
  bool mem_type[2] = {true, false};
  bool copy_dir[2] = {true, false};
  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 2; ++j) {
      elapsed_time = cuda_host_alloc_test(SIZE, mem_type[i], copy_dir[j]);
      print_time(MB, elapsed_time, mem_type[i], copy_dir[j]);
    }
  }
  return 0;
}
